#include "hip/hip_runtime.h"
#include "functions.cuh"
#include "segmenter.cuh"
#include <time.h>
#include "weights.cuh"

int main(int argc, char *argv[]){
  /*FILE *output;

  if((output = fopen("cuda_res.txt","w"))==NULL){
      printf("Failed opening files\n");
      return 1;
  }*/

  //datatype value;
  FILE* f0;

  if((f0 = fopen("test_data.bin", "rb"))==NULL){
      printf("Failed opening files\n");
      return 1;
  }
    
  datatype *test_data;
  datatype *y;
  hipHostMalloc((void**)&test_data,sizeof(datatype) *N_FEATURES*N);
  hipHostMalloc((void**)&y,sizeof(datatype) *N*N_STATES);

  fread(test_data, sizeof(datatype), N_FEATURES*N, f0);

  //hipProfilerStart();
  clock_t time1 = clock();

  //Weights
  datatype *d_x, *d_y;
  datatype *d_enc_0_conv_relu_0_w, *d_enc_0_conv_relu_1_w, *d_enc_1_conv_relu_0_w, *d_enc_1_conv_relu_1_w, *d_enc_2_conv_relu_0_w,
          *d_enc_2_conv_relu_1_w, *d_enc_3_conv_relu_0_w, *d_enc_3_conv_relu_1_w, *d_central_conv_relu_0_w, *d_central_conv_relu_1_w,
          *d_dec_0_up_conv_relu_w, *d_dec_0_conv_relu_0_w, *d_dec_0_conv_relu_1_w, *d_dec_1_up_conv_relu_w, *d_dec_1_conv_relu_0_w,
          *d_dec_1_conv_relu_1_w, *d_dec_2_up_conv_relu_w, *d_dec_2_conv_relu_0_w, *d_dec_2_conv_relu_1_w, *d_dec_3_up_conv_relu_w,
          *d_dec_3_conv_relu_0_w, *d_dec_3_conv_relu_1_w, *d_final_conv_w;

  //Feature maps
  datatype *d_enc_0_conv_relu_0, *d_enc_0_conv_relu_1, *d_enc_0_maxpool;
  datatype *d_enc_1_conv_relu_0, *d_enc_1_conv_relu_1, *d_enc_1_maxpool;
  datatype *d_enc_2_conv_relu_0, *d_enc_2_conv_relu_1, *d_enc_2_maxpool;
  datatype *d_enc_3_conv_relu_0, *d_enc_3_conv_relu_1, *d_enc_3_maxpool;
  datatype *d_central_conv_relu_0, *d_central_conv_relu_1;
  datatype *d_dec_0_upsample, *d_dec_0_up_conv_relu, *d_dec_0_concatenate, *d_dec_0_conv_relu_0, *d_dec_0_conv_relu_1; 
  datatype *d_dec_1_upsample, *d_dec_1_up_conv_relu, *d_dec_1_concatenate, *d_dec_1_conv_relu_0, *d_dec_1_conv_relu_1;
  datatype *d_dec_2_upsample, *d_dec_2_up_conv_relu, *d_dec_2_concatenate, *d_dec_2_conv_relu_0, *d_dec_2_conv_relu_1;
  datatype *d_dec_3_upsample, *d_dec_3_up_conv_relu, *d_dec_3_concatenate, *d_dec_3_conv_relu_0, *d_dec_3_conv_relu_1; 
  datatype *d_final_conv;

  //Weights and feature maps allocation/initialization
  hipMalloc((void**)&d_x, sizeof(datatype) * N*N_FEATURES);
  hipMemcpyAsync(d_x, test_data, N*N_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_enc_0_conv_relu_0, sizeof(datatype) * ENC_0_CONV_RELU_0_N * ENC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_0_conv_relu_0_w, sizeof(datatype) * ENC_0_CONV_RELU_0_K * ENC_0_CONV_RELU_0_INPUT_FEATURES*ENC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_0_conv_relu_0_w, enc_0_conv_relu_0_w, ENC_0_CONV_RELU_0_K * ENC_0_CONV_RELU_0_INPUT_FEATURES*ENC_0_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_enc_0_conv_relu_1, sizeof(datatype) * ENC_0_CONV_RELU_1_N * ENC_0_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_0_conv_relu_1_w, sizeof(datatype) * ENC_0_CONV_RELU_1_K * ENC_0_CONV_RELU_1_INPUT_FEATURES*ENC_0_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_0_conv_relu_1_w, enc_0_conv_relu_1_w, ENC_0_CONV_RELU_1_K * ENC_0_CONV_RELU_1_INPUT_FEATURES*ENC_0_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_enc_0_maxpool, sizeof(datatype) * (ENC_0_CONV_RELU_1_N/2) * ENC_0_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_enc_1_conv_relu_0, sizeof(datatype) * ENC_1_CONV_RELU_0_N * ENC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_1_conv_relu_0_w, sizeof(datatype) * ENC_1_CONV_RELU_0_K*ENC_1_CONV_RELU_0_INPUT_FEATURES*ENC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_1_conv_relu_0_w, enc_1_conv_relu_0_w, ENC_1_CONV_RELU_0_K * ENC_1_CONV_RELU_0_INPUT_FEATURES*ENC_1_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_enc_1_conv_relu_1, sizeof(datatype) * ENC_1_CONV_RELU_1_N * ENC_1_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_1_conv_relu_1_w, sizeof(datatype) * ENC_1_CONV_RELU_1_K*ENC_1_CONV_RELU_1_INPUT_FEATURES*ENC_1_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_1_conv_relu_1_w, enc_1_conv_relu_1_w, ENC_1_CONV_RELU_1_K * ENC_1_CONV_RELU_1_INPUT_FEATURES*ENC_1_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_enc_1_maxpool, sizeof(datatype) * (ENC_1_CONV_RELU_1_N/2) * ENC_1_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_enc_2_conv_relu_0, sizeof(datatype) * ENC_2_CONV_RELU_0_N * ENC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_2_conv_relu_0_w, sizeof(datatype) * ENC_2_CONV_RELU_0_K*ENC_2_CONV_RELU_0_INPUT_FEATURES*ENC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_2_conv_relu_0_w, enc_2_conv_relu_0_w, ENC_2_CONV_RELU_0_K * ENC_2_CONV_RELU_0_INPUT_FEATURES*ENC_2_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_enc_2_conv_relu_1, sizeof(datatype) * ENC_2_CONV_RELU_1_N * ENC_2_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_2_conv_relu_1_w, sizeof(datatype) * ENC_2_CONV_RELU_1_K*ENC_2_CONV_RELU_1_INPUT_FEATURES*ENC_2_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_2_conv_relu_1_w, enc_2_conv_relu_1_w, ENC_2_CONV_RELU_1_K * ENC_2_CONV_RELU_1_INPUT_FEATURES*ENC_2_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);    
  hipMalloc((void**)&d_enc_2_maxpool, sizeof(datatype) * (ENC_2_CONV_RELU_1_N/2) * ENC_2_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_enc_3_conv_relu_0, sizeof(datatype) * ENC_3_CONV_RELU_0_N * ENC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_3_conv_relu_0_w, sizeof(datatype) * ENC_3_CONV_RELU_0_K*ENC_3_CONV_RELU_0_INPUT_FEATURES*ENC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_3_conv_relu_0_w, enc_3_conv_relu_0_w, ENC_3_CONV_RELU_0_K * ENC_3_CONV_RELU_0_INPUT_FEATURES*ENC_3_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_enc_3_conv_relu_1, sizeof(datatype) * ENC_3_CONV_RELU_1_N * ENC_3_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_3_conv_relu_1_w, sizeof(datatype) * ENC_3_CONV_RELU_1_K*ENC_3_CONV_RELU_1_INPUT_FEATURES*ENC_3_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_enc_3_conv_relu_1_w, enc_3_conv_relu_1_w, ENC_3_CONV_RELU_1_K * ENC_3_CONV_RELU_1_INPUT_FEATURES*ENC_3_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);    
  hipMalloc((void**)&d_enc_3_maxpool, sizeof(datatype) * (ENC_3_CONV_RELU_1_N/2) * ENC_3_CONV_RELU_1_OUTPUT_FEATURES);
  
  hipMalloc((void**)&d_central_conv_relu_0, sizeof(datatype) * CENTRAL_CONV_RELU_0_N * CENTRAL_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_central_conv_relu_0_w, sizeof(datatype) * CENTRAL_CONV_RELU_0_K*CENTRAL_CONV_RELU_0_INPUT_FEATURES*CENTRAL_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_central_conv_relu_0_w, central_conv_relu_0_w, CENTRAL_CONV_RELU_0_K*CENTRAL_CONV_RELU_0_INPUT_FEATURES*CENTRAL_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_central_conv_relu_1, sizeof(datatype) * CENTRAL_CONV_RELU_1_N * CENTRAL_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_central_conv_relu_1_w, sizeof(datatype) * CENTRAL_CONV_RELU_1_K*CENTRAL_CONV_RELU_1_INPUT_FEATURES*CENTRAL_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_central_conv_relu_1_w, central_conv_relu_1_w, CENTRAL_CONV_RELU_1_K*CENTRAL_CONV_RELU_1_INPUT_FEATURES*CENTRAL_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_0_upsample, sizeof(datatype) * DEC_0_UP_CONV_RELU_N * DEC_0_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_up_conv_relu, sizeof(datatype) * DEC_0_UP_CONV_RELU_N * DEC_0_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_up_conv_relu_w, sizeof(datatype) * DEC_0_UP_CONV_RELU_K*DEC_0_UP_CONV_RELU_INPUT_FEATURES*DEC_0_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_0_up_conv_relu_w, dec_0_up_conv_relu_w, DEC_0_UP_CONV_RELU_K*DEC_0_UP_CONV_RELU_INPUT_FEATURES*DEC_0_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_dec_0_concatenate, sizeof(datatype) * DEC_0_UP_CONV_RELU_N * DEC_0_UP_CONV_RELU_OUTPUT_FEATURES*2);

  hipMalloc((void**)&d_dec_0_conv_relu_0, sizeof(datatype) * DEC_0_CONV_RELU_0_N * DEC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_conv_relu_0_w, sizeof(datatype) * DEC_0_CONV_RELU_0_K*DEC_0_CONV_RELU_0_INPUT_FEATURES*DEC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_0_conv_relu_0_w, dec_0_conv_relu_0_w, DEC_0_CONV_RELU_0_K*DEC_0_CONV_RELU_0_INPUT_FEATURES*DEC_0_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_0_conv_relu_1, sizeof(datatype) * DEC_0_CONV_RELU_1_N * DEC_0_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_conv_relu_1_w, sizeof(datatype) * DEC_0_CONV_RELU_1_K*DEC_0_CONV_RELU_1_INPUT_FEATURES*DEC_0_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_0_conv_relu_1_w, dec_0_conv_relu_1_w, DEC_0_CONV_RELU_1_K*DEC_0_CONV_RELU_1_INPUT_FEATURES*DEC_0_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_1_upsample, sizeof(datatype) * DEC_1_UP_CONV_RELU_N * DEC_1_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_up_conv_relu, sizeof(datatype) * DEC_1_UP_CONV_RELU_N * DEC_1_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_up_conv_relu_w, sizeof(datatype) * DEC_1_UP_CONV_RELU_K*DEC_1_UP_CONV_RELU_INPUT_FEATURES*DEC_1_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_1_up_conv_relu_w, dec_1_up_conv_relu_w, DEC_1_UP_CONV_RELU_K*DEC_1_UP_CONV_RELU_INPUT_FEATURES*DEC_1_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_dec_1_concatenate, sizeof(datatype) * DEC_1_UP_CONV_RELU_N * DEC_1_UP_CONV_RELU_OUTPUT_FEATURES*2);

  hipMalloc((void**)&d_dec_1_conv_relu_0, sizeof(datatype) * DEC_1_CONV_RELU_0_N * DEC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_conv_relu_0_w, sizeof(datatype) * DEC_1_CONV_RELU_0_K*DEC_1_CONV_RELU_0_INPUT_FEATURES*DEC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_1_conv_relu_0_w, dec_1_conv_relu_0_w, DEC_1_CONV_RELU_0_K*DEC_1_CONV_RELU_0_INPUT_FEATURES*DEC_1_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_1_conv_relu_1, sizeof(datatype) * DEC_1_CONV_RELU_1_N * DEC_1_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_conv_relu_1_w, sizeof(datatype) * DEC_1_CONV_RELU_1_K*DEC_1_CONV_RELU_1_INPUT_FEATURES*DEC_1_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_1_conv_relu_1_w, dec_1_conv_relu_1_w, DEC_1_CONV_RELU_1_K*DEC_1_CONV_RELU_1_INPUT_FEATURES*DEC_1_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_2_upsample, sizeof(datatype) * DEC_2_UP_CONV_RELU_N * DEC_2_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_up_conv_relu, sizeof(datatype) * DEC_2_UP_CONV_RELU_N * DEC_2_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_up_conv_relu_w, sizeof(datatype) * DEC_2_UP_CONV_RELU_K*DEC_2_UP_CONV_RELU_INPUT_FEATURES*DEC_2_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_2_up_conv_relu_w, dec_2_up_conv_relu_w, DEC_2_UP_CONV_RELU_K*DEC_2_UP_CONV_RELU_INPUT_FEATURES*DEC_2_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_dec_2_concatenate, sizeof(datatype) * DEC_2_UP_CONV_RELU_N * DEC_2_UP_CONV_RELU_OUTPUT_FEATURES*2);

  hipMalloc((void**)&d_dec_2_conv_relu_0, sizeof(datatype) * DEC_2_CONV_RELU_0_N * DEC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_conv_relu_0_w, sizeof(datatype) * DEC_2_CONV_RELU_0_K*DEC_2_CONV_RELU_0_INPUT_FEATURES*DEC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_2_conv_relu_0_w, dec_2_conv_relu_0_w, DEC_2_CONV_RELU_0_K*DEC_2_CONV_RELU_0_INPUT_FEATURES*DEC_2_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_2_conv_relu_1, sizeof(datatype) * DEC_2_CONV_RELU_1_N * DEC_2_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_conv_relu_1_w, sizeof(datatype) * DEC_2_CONV_RELU_1_K*DEC_2_CONV_RELU_1_INPUT_FEATURES*DEC_2_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_2_conv_relu_1_w, dec_2_conv_relu_1_w, DEC_2_CONV_RELU_1_K*DEC_2_CONV_RELU_1_INPUT_FEATURES*DEC_2_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_3_upsample, sizeof(datatype) * DEC_3_UP_CONV_RELU_N * DEC_3_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_up_conv_relu, sizeof(datatype) * DEC_3_UP_CONV_RELU_N * DEC_3_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_up_conv_relu_w, sizeof(datatype) * DEC_3_UP_CONV_RELU_K*DEC_3_UP_CONV_RELU_INPUT_FEATURES*DEC_3_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_3_up_conv_relu_w, dec_3_up_conv_relu_w, DEC_3_UP_CONV_RELU_K*DEC_3_UP_CONV_RELU_INPUT_FEATURES*DEC_3_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);
  hipMalloc((void**)&d_dec_3_concatenate, sizeof(datatype) * DEC_3_UP_CONV_RELU_N * DEC_3_UP_CONV_RELU_OUTPUT_FEATURES*2);

  hipMalloc((void**)&d_dec_3_conv_relu_0, sizeof(datatype) * DEC_3_CONV_RELU_0_N * DEC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_conv_relu_0_w, sizeof(datatype) * DEC_3_CONV_RELU_0_K*DEC_3_CONV_RELU_0_INPUT_FEATURES*DEC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_3_conv_relu_0_w, dec_3_conv_relu_0_w, DEC_3_CONV_RELU_0_K*DEC_3_CONV_RELU_0_INPUT_FEATURES*DEC_3_CONV_RELU_0_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_dec_3_conv_relu_1, sizeof(datatype) * DEC_3_CONV_RELU_1_N * DEC_3_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_conv_relu_1_w, sizeof(datatype) * DEC_3_CONV_RELU_1_K*DEC_3_CONV_RELU_1_INPUT_FEATURES*DEC_3_CONV_RELU_1_OUTPUT_FEATURES);
  hipMemcpyAsync(d_dec_3_conv_relu_1_w, dec_3_conv_relu_1_w, DEC_3_CONV_RELU_1_K*DEC_3_CONV_RELU_1_INPUT_FEATURES*DEC_3_CONV_RELU_1_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_final_conv_w, sizeof(datatype) * FINAL_CONV_K*FINAL_CONV_INPUT_FEATURES*FINAL_CONV_OUTPUT_FEATURES);
  hipMemcpyAsync(d_final_conv_w, final_conv_w, FINAL_CONV_K*FINAL_CONV_INPUT_FEATURES*FINAL_CONV_OUTPUT_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

  hipMalloc((void**)&d_y, sizeof(datatype) * N*N_STATES);
  hipMalloc((void**)&d_final_conv, sizeof(datatype) * FINAL_CONV_N * FINAL_CONV_OUTPUT_FEATURES);
  hipMemsetAsync(d_final_conv, 0, sizeof(datatype)* FINAL_CONV_N * FINAL_CONV_OUTPUT_FEATURES);
  //checkCudaError(__LINE__);

  dim3 dimBlock(THREADS, THREADS); //each block is THREADxTHREAD
	dim3 dimGrid_enc00((ENC_0_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_0_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc01((ENC_0_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_0_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max01((ENC_0_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_0_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_enc10((ENC_1_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_1_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc11((ENC_1_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_1_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max11((ENC_1_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_1_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_enc20((ENC_2_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_2_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc21((ENC_2_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_2_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max21((ENC_2_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_2_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_enc30((ENC_3_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_3_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc31((ENC_3_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_3_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max31((ENC_3_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_3_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_cent0((CENTRAL_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (CENTRAL_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_cent1((CENTRAL_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (CENTRAL_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_0_up((DEC_0_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_0_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_0_up_conv((DEC_0_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_0_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_0_conc((DEC_0_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_0_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_00((DEC_0_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_0_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_01((DEC_0_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_0_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_1_up((DEC_1_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_1_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_1_up_conv((DEC_1_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_1_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_1_conc((DEC_1_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_1_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_10((DEC_1_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_1_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_11((DEC_1_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_1_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_2_up((DEC_2_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_2_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_2_up_conv((DEC_2_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_2_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_2_conc((DEC_2_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_2_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_20((DEC_2_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_2_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_21((DEC_2_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_2_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_3_up((DEC_3_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_3_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_3_up_conv((DEC_3_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_3_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_3_conc((DEC_3_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_3_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_30((DEC_3_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_3_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_31((DEC_3_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_3_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_fin((FINAL_CONV_N+dimBlock.x-1)/dimBlock.x, (FINAL_CONV_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  hipDeviceSynchronize();
  //checkCudaError(__LINE__);

  clock_t time2 = clock();
  
  int byte_read;
  int i=0;
  
  do{  
    //-----------------------------ENCODER 0--------------------------------------
    conv_relu<<<dimGrid_enc00, dimBlock>>>(ENC_0_CONV_RELU_0_OUTPUT_FEATURES,ENC_0_CONV_RELU_0_N,ENC_0_CONV_RELU_0_K,ENC_0_CONV_RELU_0_INPUT_FEATURES,d_enc_0_conv_relu_0_w,d_x,d_enc_0_conv_relu_0);
    conv_relu<<<dimGrid_enc01, dimBlock>>>(ENC_0_CONV_RELU_1_OUTPUT_FEATURES,ENC_0_CONV_RELU_1_N,ENC_0_CONV_RELU_1_K,ENC_0_CONV_RELU_1_INPUT_FEATURES,d_enc_0_conv_relu_1_w,d_enc_0_conv_relu_0,d_enc_0_conv_relu_1);
    maxpooling<<<dimGrid_max01, dimBlock,sizeof(datatype) * 2*ENC_0_CONV_RELU_1_N * ENC_0_CONV_RELU_1_OUTPUT_FEATURES>>>(ENC_0_CONV_RELU_1_OUTPUT_FEATURES, ENC_0_CONV_RELU_1_N, d_enc_0_maxpool, d_enc_0_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------
    
    byte_read=fread(test_data, sizeof(datatype), N_FEATURES*N, f0);
    hipMemcpyAsync(d_x, test_data, N*N_FEATURES * sizeof(datatype), hipMemcpyHostToDevice);

    //-----------------------------ENCODER 1--------------------------------------
    conv_relu<<<dimGrid_enc10, dimBlock>>>(ENC_1_CONV_RELU_0_OUTPUT_FEATURES,ENC_1_CONV_RELU_0_N,ENC_1_CONV_RELU_0_K,ENC_1_CONV_RELU_0_INPUT_FEATURES,d_enc_1_conv_relu_0_w,d_enc_0_maxpool,d_enc_1_conv_relu_0);
    conv_relu<<<dimGrid_enc11, dimBlock>>>(ENC_1_CONV_RELU_1_OUTPUT_FEATURES,ENC_1_CONV_RELU_1_N,ENC_1_CONV_RELU_1_K,ENC_1_CONV_RELU_1_INPUT_FEATURES,d_enc_1_conv_relu_1_w,d_enc_1_conv_relu_0,d_enc_1_conv_relu_1);
    maxpooling<<<dimGrid_max11, dimBlock,sizeof(datatype) * 2* ENC_1_CONV_RELU_1_N * ENC_1_CONV_RELU_1_OUTPUT_FEATURES>>>(ENC_1_CONV_RELU_1_OUTPUT_FEATURES, ENC_1_CONV_RELU_1_N, d_enc_1_maxpool, d_enc_1_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------
    
    if(i>0){
      hipMemcpyAsync(y, d_y, N*N_STATES * sizeof(datatype), hipMemcpyDeviceToHost);
    }
    
    //-----------------------------ENCODER 2--------------------------------------
    conv_relu<<<dimGrid_enc20, dimBlock>>>(ENC_2_CONV_RELU_0_OUTPUT_FEATURES,ENC_2_CONV_RELU_0_N,ENC_2_CONV_RELU_0_K,ENC_2_CONV_RELU_0_INPUT_FEATURES,d_enc_2_conv_relu_0_w,d_enc_1_maxpool,d_enc_2_conv_relu_0);
    conv_relu<<<dimGrid_enc21, dimBlock>>>(ENC_2_CONV_RELU_1_OUTPUT_FEATURES,ENC_2_CONV_RELU_1_N,ENC_2_CONV_RELU_1_K,ENC_2_CONV_RELU_1_INPUT_FEATURES,d_enc_2_conv_relu_1_w,d_enc_2_conv_relu_0,d_enc_2_conv_relu_1);
    maxpooling<<<dimGrid_max21, dimBlock,sizeof(datatype) * 2* ENC_2_CONV_RELU_1_N * ENC_2_CONV_RELU_1_OUTPUT_FEATURES>>>(ENC_2_CONV_RELU_1_OUTPUT_FEATURES, ENC_2_CONV_RELU_1_N, d_enc_2_maxpool, d_enc_2_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------ENCODER 3--------------------------------------
    conv_relu<<<dimGrid_enc30, dimBlock>>>(ENC_3_CONV_RELU_0_OUTPUT_FEATURES,ENC_3_CONV_RELU_0_N,ENC_3_CONV_RELU_0_K,ENC_3_CONV_RELU_0_INPUT_FEATURES,d_enc_3_conv_relu_0_w,d_enc_2_maxpool,d_enc_3_conv_relu_0);
    conv_relu<<<dimGrid_enc31, dimBlock>>>(ENC_3_CONV_RELU_1_OUTPUT_FEATURES,ENC_3_CONV_RELU_1_N,ENC_3_CONV_RELU_1_K,ENC_3_CONV_RELU_1_INPUT_FEATURES,d_enc_3_conv_relu_1_w,d_enc_3_conv_relu_0,d_enc_3_conv_relu_1);
    maxpooling<<<dimGrid_max31, dimBlock,sizeof(datatype) * 2*ENC_3_CONV_RELU_1_N * ENC_3_CONV_RELU_1_OUTPUT_FEATURES>>>(ENC_3_CONV_RELU_1_OUTPUT_FEATURES, ENC_3_CONV_RELU_1_N, d_enc_3_maxpool, d_enc_3_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //--------------------------CENTRAL PART--------------------------------------
    conv_relu<<<dimGrid_cent0, dimBlock>>>(CENTRAL_CONV_RELU_0_OUTPUT_FEATURES,CENTRAL_CONV_RELU_0_N,CENTRAL_CONV_RELU_0_K,CENTRAL_CONV_RELU_0_INPUT_FEATURES,d_central_conv_relu_0_w,d_enc_3_maxpool,d_central_conv_relu_0);
    conv_relu<<<dimGrid_cent1, dimBlock>>>(CENTRAL_CONV_RELU_1_OUTPUT_FEATURES,CENTRAL_CONV_RELU_1_N,CENTRAL_CONV_RELU_1_K,CENTRAL_CONV_RELU_1_INPUT_FEATURES,d_central_conv_relu_1_w,d_central_conv_relu_0,d_central_conv_relu_1); 
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 0--------------------------------------
    upsampling<<<dimGrid_dec_0_up, dimBlock,sizeof(datatype) * CENTRAL_CONV_RELU_1_N * CENTRAL_CONV_RELU_1_OUTPUT_FEATURES>>>(DEC_0_UP_CONV_RELU_INPUT_FEATURES, DEC_0_UP_CONV_RELU_N,CENTRAL_CONV_RELU_1_OUTPUT_FEATURES, d_dec_0_upsample, d_central_conv_relu_1);
    conv_relu<<<dimGrid_dec_0_up_conv, dimBlock>>>(DEC_0_UP_CONV_RELU_OUTPUT_FEATURES,DEC_0_UP_CONV_RELU_N,DEC_0_UP_CONV_RELU_K,DEC_0_UP_CONV_RELU_INPUT_FEATURES,d_dec_0_up_conv_relu_w,d_dec_0_upsample,d_dec_0_up_conv_relu);
    concatenation<<<dimGrid_dec_0_conc, dimBlock>>>(DEC_0_UP_CONV_RELU_OUTPUT_FEATURES,DEC_0_UP_CONV_RELU_N, d_dec_0_concatenate, d_enc_3_conv_relu_1,d_dec_0_up_conv_relu);
    conv_relu<<<dimGrid_dec_00, dimBlock>>>(DEC_0_CONV_RELU_0_OUTPUT_FEATURES,DEC_0_CONV_RELU_0_N,DEC_0_CONV_RELU_0_K,DEC_0_CONV_RELU_0_INPUT_FEATURES,d_dec_0_conv_relu_0_w,d_dec_0_concatenate,d_dec_0_conv_relu_0);
    conv_relu<<<dimGrid_dec_01, dimBlock>>>(DEC_0_CONV_RELU_1_OUTPUT_FEATURES,DEC_0_CONV_RELU_1_N,DEC_0_CONV_RELU_1_K,DEC_0_CONV_RELU_1_INPUT_FEATURES,d_dec_0_conv_relu_1_w,d_dec_0_conv_relu_0,d_dec_0_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 1--------------------------------------
    upsampling<<<dimGrid_dec_1_up, dimBlock,sizeof(datatype) * DEC_0_CONV_RELU_1_N * DEC_0_CONV_RELU_1_OUTPUT_FEATURES>>>(DEC_1_UP_CONV_RELU_INPUT_FEATURES, DEC_1_UP_CONV_RELU_N, DEC_0_CONV_RELU_1_OUTPUT_FEATURES, d_dec_1_upsample, d_dec_0_conv_relu_1);
    conv_relu<<<dimGrid_dec_1_up_conv, dimBlock>>>(DEC_1_UP_CONV_RELU_OUTPUT_FEATURES,DEC_1_UP_CONV_RELU_N,DEC_1_UP_CONV_RELU_K,DEC_1_UP_CONV_RELU_INPUT_FEATURES,d_dec_1_up_conv_relu_w,d_dec_1_upsample,d_dec_1_up_conv_relu);
    concatenation<<<dimGrid_dec_1_conc, dimBlock>>>(DEC_1_UP_CONV_RELU_OUTPUT_FEATURES,DEC_1_UP_CONV_RELU_N, d_dec_1_concatenate, d_enc_2_conv_relu_1,d_dec_1_up_conv_relu);
    conv_relu<<<dimGrid_dec_10, dimBlock>>>(DEC_1_CONV_RELU_0_OUTPUT_FEATURES,DEC_1_CONV_RELU_0_N,DEC_1_CONV_RELU_0_K,DEC_1_CONV_RELU_0_INPUT_FEATURES,d_dec_1_conv_relu_0_w,d_dec_1_concatenate,d_dec_1_conv_relu_0);
    conv_relu<<<dimGrid_dec_11, dimBlock>>>(DEC_1_CONV_RELU_1_OUTPUT_FEATURES,DEC_1_CONV_RELU_1_N,DEC_1_CONV_RELU_1_K,DEC_1_CONV_RELU_1_INPUT_FEATURES,d_dec_1_conv_relu_1_w,d_dec_1_conv_relu_0,d_dec_1_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 2--------------------------------------
    upsampling<<<dimGrid_dec_2_up, dimBlock,sizeof(datatype) * DEC_1_CONV_RELU_1_N * DEC_1_CONV_RELU_1_OUTPUT_FEATURES>>>(DEC_2_UP_CONV_RELU_INPUT_FEATURES, DEC_2_UP_CONV_RELU_N, DEC_1_CONV_RELU_1_OUTPUT_FEATURES, d_dec_2_upsample, d_dec_1_conv_relu_1);
    conv_relu<<<dimGrid_dec_2_up_conv, dimBlock>>>(DEC_2_UP_CONV_RELU_OUTPUT_FEATURES,DEC_2_UP_CONV_RELU_N,DEC_2_UP_CONV_RELU_K,DEC_2_UP_CONV_RELU_INPUT_FEATURES,d_dec_2_up_conv_relu_w,d_dec_2_upsample,d_dec_2_up_conv_relu);
    concatenation<<<dimGrid_dec_2_conc, dimBlock>>>(DEC_2_UP_CONV_RELU_OUTPUT_FEATURES,DEC_2_UP_CONV_RELU_N, d_dec_2_concatenate, d_enc_1_conv_relu_1,d_dec_2_up_conv_relu);
    conv_relu<<<dimGrid_dec_20, dimBlock>>>(DEC_2_CONV_RELU_0_OUTPUT_FEATURES,DEC_2_CONV_RELU_0_N,DEC_2_CONV_RELU_0_K,DEC_2_CONV_RELU_0_INPUT_FEATURES,d_dec_2_conv_relu_0_w,d_dec_2_concatenate,d_dec_2_conv_relu_0);
    conv_relu<<<dimGrid_dec_21, dimBlock>>>(DEC_2_CONV_RELU_1_OUTPUT_FEATURES,DEC_2_CONV_RELU_1_N,DEC_2_CONV_RELU_1_K,DEC_2_CONV_RELU_1_INPUT_FEATURES,d_dec_2_conv_relu_1_w,d_dec_2_conv_relu_0,d_dec_2_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 3--------------------------------------
    upsampling<<<dimGrid_dec_3_up, dimBlock,sizeof(datatype) * DEC_2_CONV_RELU_1_N * DEC_2_CONV_RELU_1_OUTPUT_FEATURES>>>(DEC_3_UP_CONV_RELU_INPUT_FEATURES, DEC_3_UP_CONV_RELU_N, DEC_2_CONV_RELU_1_OUTPUT_FEATURES, d_dec_3_upsample, d_dec_2_conv_relu_1);
    conv_relu<<<dimGrid_dec_3_up_conv, dimBlock>>>(DEC_3_UP_CONV_RELU_OUTPUT_FEATURES,DEC_3_UP_CONV_RELU_N,DEC_3_UP_CONV_RELU_K,DEC_3_UP_CONV_RELU_INPUT_FEATURES,d_dec_3_up_conv_relu_w,d_dec_3_upsample,d_dec_3_up_conv_relu);
    concatenation<<<dimGrid_dec_3_conc, dimBlock>>>(DEC_3_UP_CONV_RELU_OUTPUT_FEATURES,DEC_3_UP_CONV_RELU_N, d_dec_3_concatenate, d_enc_0_conv_relu_1,d_dec_3_up_conv_relu);
    conv_relu<<<dimGrid_dec_30, dimBlock>>>(DEC_3_CONV_RELU_0_OUTPUT_FEATURES,DEC_3_CONV_RELU_0_N,DEC_3_CONV_RELU_0_K,DEC_3_CONV_RELU_0_INPUT_FEATURES,d_dec_3_conv_relu_0_w,d_dec_3_concatenate,d_dec_3_conv_relu_0);
    conv_relu<<<dimGrid_dec_31, dimBlock>>>(DEC_3_CONV_RELU_1_OUTPUT_FEATURES,DEC_3_CONV_RELU_1_N,DEC_3_CONV_RELU_1_K,DEC_3_CONV_RELU_1_INPUT_FEATURES,d_dec_3_conv_relu_1_w,d_dec_3_conv_relu_0,d_dec_3_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //----------------------------FINAL LAYER--------------------------------------
    conv_relu_last_layer<<<dimGrid_fin, dimBlock>>>(FINAL_CONV_OUTPUT_FEATURES,FINAL_CONV_N,FINAL_CONV_K,FINAL_CONV_INPUT_FEATURES,d_final_conv_w,d_dec_3_conv_relu_1,d_final_conv);
    Softmax<<<1,FINAL_CONV_N>>>(d_final_conv, d_y);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    if(i%7500==0){ //print elapsed time at each 30 patients
    	hipDeviceSynchronize();
	    printf("Processed %d samples - time: %.5f seconds\n", (i+1)/250, ((double)clock()-time1) / CLOCKS_PER_SEC);
    }

    
    //VALIDATION
    /*hipMemcpy(y, d_y, N*N_STATES * sizeof(datatype), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for(int j=0; j<N; j++){
      for(int k=0; k<N_STATES; k++){
        fprintf(output,"%.10f ", y[j*N_STATES+k]);
      }
      fprintf(output,"\n");
    }*/
        
    i++;
  }while(byte_read== N_FEATURES*N);
  hipDeviceSynchronize();
  
  time2 = clock()-time2;

  hipMemcpy(y, d_y, N*N_STATES * sizeof(datatype), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  //checkCudaError(__LINE__);

  time1 = clock()-time1;
	printf("Time required for memory and parameters initialization: %.5f seconds\n", ((double)time1-time2) / CLOCKS_PER_SEC);
  printf("Elapsed computation time: %.5f seconds\n", ((double)time2) / CLOCKS_PER_SEC);
  printf("\nTotale elapsed time: %.5f seconds\n\n", ((double)(time1)) / CLOCKS_PER_SEC);


  //fclose(output);
  fclose(f0);

  hipFree(d_enc_0_conv_relu_0);
  hipFree(d_enc_0_conv_relu_1);
  hipFree(d_enc_0_maxpool);

  hipFree(d_enc_1_conv_relu_0);
  hipFree(d_enc_1_conv_relu_1);
  hipFree(d_enc_1_maxpool);

  hipFree(d_enc_2_conv_relu_0);
  hipFree(d_enc_2_conv_relu_1);
  hipFree(d_enc_2_maxpool);

  hipFree(d_enc_3_conv_relu_0);
  hipFree(d_enc_3_conv_relu_1);
  hipFree(d_enc_3_maxpool);

  hipFree(d_central_conv_relu_0);
  hipFree(d_central_conv_relu_1);

  hipFree(d_dec_0_upsample);
  hipFree(d_dec_0_up_conv_relu);
  hipFree(d_dec_0_concatenate);
  hipFree(d_dec_0_conv_relu_0);
  hipFree(d_dec_0_conv_relu_1);

  hipFree(d_dec_1_upsample);
  hipFree(d_dec_1_up_conv_relu);
  hipFree(d_dec_1_concatenate);
  hipFree(d_dec_1_conv_relu_0);
  hipFree(d_dec_1_conv_relu_1);

  hipFree(d_dec_2_upsample);
  hipFree(d_dec_2_up_conv_relu);
  hipFree(d_dec_2_concatenate);
  hipFree(d_dec_2_conv_relu_0);
  hipFree(d_dec_2_conv_relu_1);

  hipFree(d_dec_3_upsample);
  hipFree(d_dec_3_up_conv_relu);
  hipFree(d_dec_3_concatenate);
  hipFree(d_dec_3_conv_relu_0);
  hipFree(d_dec_3_conv_relu_1);

  hipFree(d_final_conv);

  hipFree(d_x);
  hipFree(d_enc_0_conv_relu_0_w);
  hipFree(d_enc_0_conv_relu_1_w);

  hipFree(d_enc_1_conv_relu_0_w);
  hipFree(d_enc_1_conv_relu_1_w);

  hipFree(d_enc_2_conv_relu_0_w);
  hipFree(d_enc_2_conv_relu_1_w);

  hipFree(d_enc_3_conv_relu_0_w);
  hipFree(d_enc_3_conv_relu_1_w);

  hipFree(d_central_conv_relu_0_w);
  hipFree(d_central_conv_relu_1_w);

  hipFree(d_dec_0_up_conv_relu_w);
  hipFree(d_dec_0_conv_relu_0_w);
  hipFree(d_dec_0_conv_relu_1_w);

  hipFree(d_dec_1_up_conv_relu_w);
  hipFree(d_dec_1_conv_relu_0_w);
  hipFree(d_dec_1_conv_relu_1_w);

  hipFree(d_dec_2_up_conv_relu_w);
  hipFree(d_dec_2_conv_relu_0_w);
  hipFree(d_dec_2_conv_relu_1_w);

  hipFree(d_dec_3_up_conv_relu_w);
  hipFree(d_dec_3_conv_relu_0_w);
  hipFree(d_dec_3_conv_relu_1_w);

  hipFree(d_final_conv_w);
  hipFree(d_y);
  //checkCudaError(__LINE__);

  free(y);
  free(test_data);

  return 0;
}