#include "hip/hip_runtime.h"
#include "functions.cuh"
#include "segmenter.cuh"
#include <time.h>
#include "test_data.cuh"
#include "weights.cuh"

int main(){
  /*FILE *pythonOutput;

  if((pythonOutput = fopen("python_result.txt","r"))==NULL){
      printf("Failed opening files\n");
      return 1;
  }

  float value;*/
  float *y = (float *)malloc(sizeof(float) *N*N_STATES);
  float *final_conv = (float *)malloc(sizeof(float) *FINAL_CONV_N * FINAL_CONV_OUTPUT_FEATURES);

  //hipProfilerStart();
  clock_t time1 = clock();

  //Weights
  float *d_x, *d_y;
  float *d_enc_0_conv_relu_0_w, *d_enc_0_conv_relu_1_w, *d_enc_1_conv_relu_0_w, *d_enc_1_conv_relu_1_w, *d_enc_2_conv_relu_0_w,
          *d_enc_2_conv_relu_1_w, *d_enc_3_conv_relu_0_w, *d_enc_3_conv_relu_1_w, *d_central_conv_relu_0_w, *d_central_conv_relu_1_w,
          *d_dec_0_up_conv_relu_w, *d_dec_0_conv_relu_0_w, *d_dec_0_conv_relu_1_w, *d_dec_1_up_conv_relu_w, *d_dec_1_conv_relu_0_w,
          *d_dec_1_conv_relu_1_w, *d_dec_2_up_conv_relu_w, *d_dec_2_conv_relu_0_w, *d_dec_2_conv_relu_1_w, *d_dec_3_up_conv_relu_w,
          *d_dec_3_conv_relu_0_w, *d_dec_3_conv_relu_1_w, *d_final_conv_w;

  //Feature maps
  float *d_enc_0_conv_relu_0, *d_enc_0_conv_relu_1, *d_enc_0_maxpool;
  float *d_enc_1_conv_relu_0, *d_enc_1_conv_relu_1, *d_enc_1_maxpool;
  float *d_enc_2_conv_relu_0, *d_enc_2_conv_relu_1, *d_enc_2_maxpool;
  float *d_enc_3_conv_relu_0, *d_enc_3_conv_relu_1, *d_enc_3_maxpool;
  float *d_central_conv_relu_0, *d_central_conv_relu_1;
  float *d_dec_0_upsample, *d_dec_0_up_conv_relu, *d_dec_0_concatenate, *d_dec_0_conv_relu_0, *d_dec_0_conv_relu_1; 
  float *d_dec_1_upsample, *d_dec_1_up_conv_relu, *d_dec_1_concatenate, *d_dec_1_conv_relu_0, *d_dec_1_conv_relu_1;
  float *d_dec_2_upsample, *d_dec_2_up_conv_relu, *d_dec_2_concatenate, *d_dec_2_conv_relu_0, *d_dec_2_conv_relu_1;
  float *d_dec_3_upsample, *d_dec_3_up_conv_relu, *d_dec_3_concatenate, *d_dec_3_conv_relu_0, *d_dec_3_conv_relu_1; 
  float *d_final_conv;

  //Weights initialization
  hipMalloc((void**)&d_x, sizeof(float) * TEST_SAMPLES_BATCH*N*N_FEATURES);
  hipMalloc((void**)&d_enc_0_conv_relu_0_w, sizeof(float) * ENC_0_CONV_RELU_0_K * ENC_0_CONV_RELU_0_INPUT_FEATURES*ENC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_0_conv_relu_1_w, sizeof(float) * ENC_0_CONV_RELU_1_K * ENC_0_CONV_RELU_1_INPUT_FEATURES*ENC_0_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_1_conv_relu_0_w, sizeof(float) * ENC_1_CONV_RELU_0_K*ENC_1_CONV_RELU_0_INPUT_FEATURES*ENC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_1_conv_relu_1_w, sizeof(float) * ENC_1_CONV_RELU_1_K*ENC_1_CONV_RELU_1_INPUT_FEATURES*ENC_1_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_2_conv_relu_0_w, sizeof(float) * ENC_2_CONV_RELU_0_K*ENC_2_CONV_RELU_0_INPUT_FEATURES*ENC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_2_conv_relu_1_w, sizeof(float) * ENC_2_CONV_RELU_1_K*ENC_2_CONV_RELU_1_INPUT_FEATURES*ENC_2_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_3_conv_relu_0_w, sizeof(float) * ENC_3_CONV_RELU_0_K*ENC_3_CONV_RELU_0_INPUT_FEATURES*ENC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_3_conv_relu_1_w, sizeof(float) * ENC_3_CONV_RELU_1_K*ENC_3_CONV_RELU_1_INPUT_FEATURES*ENC_3_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_central_conv_relu_0_w, sizeof(float) * CENTRAL_CONV_RELU_0_K*CENTRAL_CONV_RELU_0_INPUT_FEATURES*CENTRAL_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_central_conv_relu_1_w, sizeof(float) * CENTRAL_CONV_RELU_1_K*CENTRAL_CONV_RELU_1_INPUT_FEATURES*CENTRAL_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_up_conv_relu_w, sizeof(float) * DEC_0_UP_CONV_RELU_K*DEC_0_UP_CONV_RELU_INPUT_FEATURES*DEC_0_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_conv_relu_0_w, sizeof(float) * DEC_0_CONV_RELU_0_K*DEC_0_CONV_RELU_0_INPUT_FEATURES*DEC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_conv_relu_1_w, sizeof(float) * DEC_0_CONV_RELU_1_K*DEC_0_CONV_RELU_1_INPUT_FEATURES*DEC_0_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_up_conv_relu_w, sizeof(float) * DEC_1_UP_CONV_RELU_K*DEC_1_UP_CONV_RELU_INPUT_FEATURES*DEC_1_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_conv_relu_0_w, sizeof(float) * DEC_1_CONV_RELU_0_K*DEC_1_CONV_RELU_0_INPUT_FEATURES*DEC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_conv_relu_1_w, sizeof(float) * DEC_1_CONV_RELU_1_K*DEC_1_CONV_RELU_1_INPUT_FEATURES*DEC_1_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_up_conv_relu_w, sizeof(float) * DEC_2_UP_CONV_RELU_K*DEC_2_UP_CONV_RELU_INPUT_FEATURES*DEC_2_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_conv_relu_0_w, sizeof(float) * DEC_2_CONV_RELU_0_K*DEC_2_CONV_RELU_0_INPUT_FEATURES*DEC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_conv_relu_1_w, sizeof(float) * DEC_2_CONV_RELU_1_K*DEC_2_CONV_RELU_1_INPUT_FEATURES*DEC_2_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_up_conv_relu_w, sizeof(float) * DEC_3_UP_CONV_RELU_K*DEC_3_UP_CONV_RELU_INPUT_FEATURES*DEC_3_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_conv_relu_0_w, sizeof(float) * DEC_3_CONV_RELU_0_K*DEC_3_CONV_RELU_0_INPUT_FEATURES*DEC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_conv_relu_1_w, sizeof(float) * DEC_3_CONV_RELU_1_K*DEC_3_CONV_RELU_1_INPUT_FEATURES*DEC_3_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_final_conv_w, sizeof(float) * FINAL_CONV_K*FINAL_CONV_INPUT_FEATURES*FINAL_CONV_OUTPUT_FEATURES);
  hipMalloc((void**)&d_y, sizeof(float) * N*N_STATES);
  //checkCudaError(__LINE__);
  
  hipMemcpy(d_x, test_data, TEST_SAMPLES_BATCH*N*N_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_enc_0_conv_relu_0_w, enc_0_conv_relu_0_w, ENC_0_CONV_RELU_0_K * ENC_0_CONV_RELU_0_INPUT_FEATURES*ENC_0_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_enc_0_conv_relu_1_w, enc_0_conv_relu_1_w, ENC_0_CONV_RELU_1_K * ENC_0_CONV_RELU_1_INPUT_FEATURES*ENC_0_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_enc_1_conv_relu_0_w, enc_1_conv_relu_0_w, ENC_1_CONV_RELU_0_K * ENC_1_CONV_RELU_0_INPUT_FEATURES*ENC_1_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_enc_1_conv_relu_1_w, enc_1_conv_relu_1_w, ENC_1_CONV_RELU_1_K * ENC_1_CONV_RELU_1_INPUT_FEATURES*ENC_1_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_enc_2_conv_relu_0_w, enc_2_conv_relu_0_w, ENC_2_CONV_RELU_0_K * ENC_2_CONV_RELU_0_INPUT_FEATURES*ENC_2_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_enc_2_conv_relu_1_w, enc_2_conv_relu_1_w, ENC_2_CONV_RELU_1_K * ENC_2_CONV_RELU_1_INPUT_FEATURES*ENC_2_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);    
  hipMemcpy(d_enc_3_conv_relu_0_w, enc_3_conv_relu_0_w, ENC_3_CONV_RELU_0_K * ENC_3_CONV_RELU_0_INPUT_FEATURES*ENC_3_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_enc_3_conv_relu_1_w, enc_3_conv_relu_1_w, ENC_3_CONV_RELU_1_K * ENC_3_CONV_RELU_1_INPUT_FEATURES*ENC_3_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);    
  hipMemcpy(d_central_conv_relu_0_w, central_conv_relu_0_w, CENTRAL_CONV_RELU_0_K*CENTRAL_CONV_RELU_0_INPUT_FEATURES*CENTRAL_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_central_conv_relu_1_w, central_conv_relu_1_w, CENTRAL_CONV_RELU_1_K*CENTRAL_CONV_RELU_1_INPUT_FEATURES*CENTRAL_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_0_up_conv_relu_w, dec_0_up_conv_relu_w, DEC_0_UP_CONV_RELU_K*DEC_0_UP_CONV_RELU_INPUT_FEATURES*DEC_0_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_0_conv_relu_0_w, dec_0_conv_relu_0_w, DEC_0_CONV_RELU_0_K*DEC_0_CONV_RELU_0_INPUT_FEATURES*DEC_0_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_0_conv_relu_1_w, dec_0_conv_relu_1_w, DEC_0_CONV_RELU_1_K*DEC_0_CONV_RELU_1_INPUT_FEATURES*DEC_0_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_1_up_conv_relu_w, dec_1_up_conv_relu_w, DEC_1_UP_CONV_RELU_K*DEC_1_UP_CONV_RELU_INPUT_FEATURES*DEC_1_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_1_conv_relu_0_w, dec_1_conv_relu_0_w, DEC_1_CONV_RELU_0_K*DEC_1_CONV_RELU_0_INPUT_FEATURES*DEC_1_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_1_conv_relu_1_w, dec_1_conv_relu_1_w, DEC_1_CONV_RELU_1_K*DEC_1_CONV_RELU_1_INPUT_FEATURES*DEC_1_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_2_up_conv_relu_w, dec_2_up_conv_relu_w, DEC_2_UP_CONV_RELU_K*DEC_2_UP_CONV_RELU_INPUT_FEATURES*DEC_2_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_2_conv_relu_0_w, dec_2_conv_relu_0_w, DEC_2_CONV_RELU_0_K*DEC_2_CONV_RELU_0_INPUT_FEATURES*DEC_2_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_2_conv_relu_1_w, dec_2_conv_relu_1_w, DEC_2_CONV_RELU_1_K*DEC_2_CONV_RELU_1_INPUT_FEATURES*DEC_2_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_3_up_conv_relu_w, dec_3_up_conv_relu_w, DEC_3_UP_CONV_RELU_K*DEC_3_UP_CONV_RELU_INPUT_FEATURES*DEC_3_UP_CONV_RELU_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_3_conv_relu_0_w, dec_3_conv_relu_0_w, DEC_3_CONV_RELU_0_K*DEC_3_CONV_RELU_0_INPUT_FEATURES*DEC_3_CONV_RELU_0_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_dec_3_conv_relu_1_w, dec_3_conv_relu_1_w, DEC_3_CONV_RELU_1_K*DEC_3_CONV_RELU_1_INPUT_FEATURES*DEC_3_CONV_RELU_1_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_final_conv_w, final_conv_w, FINAL_CONV_K*FINAL_CONV_INPUT_FEATURES*FINAL_CONV_OUTPUT_FEATURES * sizeof(float), hipMemcpyHostToDevice);
  //checkCudaError(__LINE__);

  //Feature maps initialization
  hipMalloc((void**)&d_enc_0_conv_relu_0, sizeof(float) * ENC_0_CONV_RELU_0_N * ENC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_0_conv_relu_1, sizeof(float) * ENC_0_CONV_RELU_1_N * ENC_0_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_0_maxpool, sizeof(float) * (ENC_0_CONV_RELU_1_N/2) * ENC_0_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_enc_1_conv_relu_0, sizeof(float) * ENC_1_CONV_RELU_0_N * ENC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_1_conv_relu_1, sizeof(float) * ENC_1_CONV_RELU_1_N * ENC_1_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_1_maxpool, sizeof(float) * (ENC_1_CONV_RELU_1_N/2) * ENC_1_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_enc_2_conv_relu_0, sizeof(float) * ENC_2_CONV_RELU_0_N * ENC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_2_conv_relu_1, sizeof(float) * ENC_2_CONV_RELU_1_N * ENC_2_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_2_maxpool, sizeof(float) * (ENC_2_CONV_RELU_1_N/2) * ENC_2_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_enc_3_conv_relu_0, sizeof(float) * ENC_3_CONV_RELU_0_N * ENC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_3_conv_relu_1, sizeof(float) * ENC_3_CONV_RELU_1_N * ENC_3_CONV_RELU_1_OUTPUT_FEATURES);
  hipMalloc((void**)&d_enc_3_maxpool, sizeof(float) * (ENC_3_CONV_RELU_1_N/2) * ENC_3_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_central_conv_relu_0, sizeof(float) * CENTRAL_CONV_RELU_0_N * CENTRAL_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_central_conv_relu_1, sizeof(float) * CENTRAL_CONV_RELU_1_N * CENTRAL_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_dec_0_upsample, sizeof(float) * DEC_0_UP_CONV_RELU_N * DEC_0_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_up_conv_relu, sizeof(float) * DEC_0_UP_CONV_RELU_N * DEC_0_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_concatenate, sizeof(float) * DEC_0_UP_CONV_RELU_N * DEC_0_UP_CONV_RELU_OUTPUT_FEATURES*2);
  hipMalloc((void**)&d_dec_0_conv_relu_0, sizeof(float) * DEC_0_CONV_RELU_0_N * DEC_0_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_0_conv_relu_1, sizeof(float) * DEC_0_CONV_RELU_1_N * DEC_0_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_dec_1_upsample, sizeof(float) * DEC_1_UP_CONV_RELU_N * DEC_1_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_up_conv_relu, sizeof(float) * DEC_1_UP_CONV_RELU_N * DEC_1_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_concatenate, sizeof(float) * DEC_1_UP_CONV_RELU_N * DEC_1_UP_CONV_RELU_OUTPUT_FEATURES*2);
  hipMalloc((void**)&d_dec_1_conv_relu_0, sizeof(float) * DEC_1_CONV_RELU_0_N * DEC_1_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_1_conv_relu_1, sizeof(float) * DEC_1_CONV_RELU_1_N * DEC_1_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_dec_2_upsample, sizeof(float) * DEC_2_UP_CONV_RELU_N * DEC_2_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_up_conv_relu, sizeof(float) * DEC_2_UP_CONV_RELU_N * DEC_2_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_concatenate, sizeof(float) * DEC_2_UP_CONV_RELU_N * DEC_2_UP_CONV_RELU_OUTPUT_FEATURES*2);
  hipMalloc((void**)&d_dec_2_conv_relu_0, sizeof(float) * DEC_2_CONV_RELU_0_N * DEC_2_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_2_conv_relu_1, sizeof(float) * DEC_2_CONV_RELU_1_N * DEC_2_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_dec_3_upsample, sizeof(float) * DEC_3_UP_CONV_RELU_N * DEC_3_UP_CONV_RELU_INPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_up_conv_relu, sizeof(float) * DEC_3_UP_CONV_RELU_N * DEC_3_UP_CONV_RELU_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_concatenate, sizeof(float) * DEC_3_UP_CONV_RELU_N * DEC_3_UP_CONV_RELU_OUTPUT_FEATURES*2);
  hipMalloc((void**)&d_dec_3_conv_relu_0, sizeof(float) * DEC_3_CONV_RELU_0_N * DEC_3_CONV_RELU_0_OUTPUT_FEATURES);
  hipMalloc((void**)&d_dec_3_conv_relu_1, sizeof(float) * DEC_3_CONV_RELU_1_N * DEC_3_CONV_RELU_1_OUTPUT_FEATURES);

  hipMalloc((void**)&d_final_conv, sizeof(float) * FINAL_CONV_N * FINAL_CONV_OUTPUT_FEATURES);
  hipMemset(d_final_conv, 0, sizeof(float)* FINAL_CONV_N * FINAL_CONV_OUTPUT_FEATURES);
  //checkCudaError(__LINE__);

  dim3 dimBlock(THREADS, THREADS); //each block is THREADxTHREAD
	dim3 dimGrid_enc00((ENC_0_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_0_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc01((ENC_0_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_0_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max01((ENC_0_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_0_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_enc10((ENC_1_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_1_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc11((ENC_1_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_1_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max11((ENC_1_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_1_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_enc20((ENC_2_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_2_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc21((ENC_2_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_2_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max21((ENC_2_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_2_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_enc30((ENC_3_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (ENC_3_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_enc31((ENC_3_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (ENC_3_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_max31((ENC_3_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((ENC_3_CONV_RELU_1_N/2)+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_cent0((CENTRAL_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (CENTRAL_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_cent1((CENTRAL_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (CENTRAL_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_0_up((DEC_0_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_0_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_0_up_conv((DEC_0_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_0_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_0_conc((DEC_0_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_0_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_00((DEC_0_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_0_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_01((DEC_0_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_0_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_1_up((DEC_1_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_1_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_1_up_conv((DEC_1_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_1_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_1_conc((DEC_1_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_1_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_10((DEC_1_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_1_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_11((DEC_1_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_1_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_2_up((DEC_2_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_2_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_2_up_conv((DEC_2_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_2_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_2_conc((DEC_2_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_2_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_20((DEC_2_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_2_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_21((DEC_2_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_2_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_dec_3_up((DEC_3_UP_CONV_RELU_INPUT_FEATURES+dimBlock.x-1)/dimBlock.x, ((DEC_3_UP_CONV_RELU_N/2)+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_3_up_conv((DEC_3_UP_CONV_RELU_N+dimBlock.x-1)/dimBlock.x, (DEC_3_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_3_conc((DEC_3_UP_CONV_RELU_OUTPUT_FEATURES+dimBlock.x-1)/dimBlock.x, (DEC_3_UP_CONV_RELU_N+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_30((DEC_3_CONV_RELU_0_N+dimBlock.x-1)/dimBlock.x, (DEC_3_CONV_RELU_0_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  dim3 dimGrid_dec_31((DEC_3_CONV_RELU_1_N+dimBlock.x-1)/dimBlock.x, (DEC_3_CONV_RELU_1_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);

  dim3 dimGrid_fin((FINAL_CONV_N+dimBlock.x-1)/dimBlock.x, (FINAL_CONV_OUTPUT_FEATURES+dimBlock.y-1)/dimBlock.y);
  //checkCudaError(__LINE__);
  
  clock_t time2 = clock();
  
  for(int i=0; i<TEST_SAMPLES_BATCH;i++){ //TEST_SAMPLES_BATCH
    //-----------------------------ENCODER 0--------------------------------------
    conv_relu<<<dimGrid_enc00, dimBlock>>>(ENC_0_CONV_RELU_0_OUTPUT_FEATURES,ENC_0_CONV_RELU_0_N,ENC_0_CONV_RELU_0_K,ENC_0_CONV_RELU_0_INPUT_FEATURES,d_enc_0_conv_relu_0_w,(d_x+i*TEST_SAMPLES_BATCH),d_enc_0_conv_relu_0);
    conv_relu<<<dimGrid_enc01, dimBlock>>>(ENC_0_CONV_RELU_1_OUTPUT_FEATURES,ENC_0_CONV_RELU_1_N,ENC_0_CONV_RELU_1_K,ENC_0_CONV_RELU_1_INPUT_FEATURES,d_enc_0_conv_relu_1_w,d_enc_0_conv_relu_0,d_enc_0_conv_relu_1);
    maxpooling<<<dimGrid_max01, dimBlock>>>(ENC_0_CONV_RELU_1_OUTPUT_FEATURES, ENC_0_CONV_RELU_1_N, d_enc_0_maxpool, d_enc_0_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------ENCODER 1--------------------------------------
    conv_relu<<<dimGrid_enc10, dimBlock>>>(ENC_1_CONV_RELU_0_OUTPUT_FEATURES,ENC_1_CONV_RELU_0_N,ENC_1_CONV_RELU_0_K,ENC_1_CONV_RELU_0_INPUT_FEATURES,d_enc_1_conv_relu_0_w,d_enc_0_maxpool,d_enc_1_conv_relu_0);
    conv_relu<<<dimGrid_enc11, dimBlock>>>(ENC_1_CONV_RELU_1_OUTPUT_FEATURES,ENC_1_CONV_RELU_1_N,ENC_1_CONV_RELU_1_K,ENC_1_CONV_RELU_1_INPUT_FEATURES,d_enc_1_conv_relu_1_w,d_enc_1_conv_relu_0,d_enc_1_conv_relu_1);
    maxpooling<<<dimGrid_max11, dimBlock>>>(ENC_1_CONV_RELU_1_OUTPUT_FEATURES, ENC_1_CONV_RELU_1_N, d_enc_1_maxpool, d_enc_1_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------ENCODER 2--------------------------------------
    conv_relu<<<dimGrid_enc20, dimBlock>>>(ENC_2_CONV_RELU_0_OUTPUT_FEATURES,ENC_2_CONV_RELU_0_N,ENC_2_CONV_RELU_0_K,ENC_2_CONV_RELU_0_INPUT_FEATURES,d_enc_2_conv_relu_0_w,d_enc_1_maxpool,d_enc_2_conv_relu_0);
    conv_relu<<<dimGrid_enc21, dimBlock>>>(ENC_2_CONV_RELU_1_OUTPUT_FEATURES,ENC_2_CONV_RELU_1_N,ENC_2_CONV_RELU_1_K,ENC_2_CONV_RELU_1_INPUT_FEATURES,d_enc_2_conv_relu_1_w,d_enc_2_conv_relu_0,d_enc_2_conv_relu_1);
    maxpooling<<<dimGrid_max21, dimBlock>>>(ENC_2_CONV_RELU_1_OUTPUT_FEATURES, ENC_2_CONV_RELU_1_N, d_enc_2_maxpool, d_enc_2_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------ENCODER 3--------------------------------------
    conv_relu<<<dimGrid_enc30, dimBlock>>>(ENC_3_CONV_RELU_0_OUTPUT_FEATURES,ENC_3_CONV_RELU_0_N,ENC_3_CONV_RELU_0_K,ENC_3_CONV_RELU_0_INPUT_FEATURES,d_enc_3_conv_relu_0_w,d_enc_2_maxpool,d_enc_3_conv_relu_0);
    conv_relu<<<dimGrid_enc31, dimBlock>>>(ENC_3_CONV_RELU_1_OUTPUT_FEATURES,ENC_3_CONV_RELU_1_N,ENC_3_CONV_RELU_1_K,ENC_3_CONV_RELU_1_INPUT_FEATURES,d_enc_3_conv_relu_1_w,d_enc_3_conv_relu_0,d_enc_3_conv_relu_1);
    maxpooling<<<dimGrid_max31, dimBlock>>>(ENC_3_CONV_RELU_1_OUTPUT_FEATURES, ENC_3_CONV_RELU_1_N, d_enc_3_maxpool, d_enc_3_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //--------------------------CENTRAL PART--------------------------------------
    conv_relu<<<dimGrid_cent0, dimBlock>>>(CENTRAL_CONV_RELU_0_OUTPUT_FEATURES,CENTRAL_CONV_RELU_0_N,CENTRAL_CONV_RELU_0_K,CENTRAL_CONV_RELU_0_INPUT_FEATURES,d_central_conv_relu_0_w,d_enc_3_maxpool,d_central_conv_relu_0);
    conv_relu<<<dimGrid_cent1, dimBlock>>>(CENTRAL_CONV_RELU_1_OUTPUT_FEATURES,CENTRAL_CONV_RELU_1_N,CENTRAL_CONV_RELU_1_K,CENTRAL_CONV_RELU_1_INPUT_FEATURES,d_central_conv_relu_1_w,d_central_conv_relu_0,d_central_conv_relu_1); 
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 0--------------------------------------
    upsampling<<<dimGrid_dec_0_up, dimBlock>>>(DEC_0_UP_CONV_RELU_INPUT_FEATURES, DEC_0_UP_CONV_RELU_N,CENTRAL_CONV_RELU_1_OUTPUT_FEATURES, d_dec_0_upsample, d_central_conv_relu_1);
    conv_relu<<<dimGrid_dec_0_up_conv, dimBlock>>>(DEC_0_UP_CONV_RELU_OUTPUT_FEATURES,DEC_0_UP_CONV_RELU_N,DEC_0_UP_CONV_RELU_K,DEC_0_UP_CONV_RELU_INPUT_FEATURES,d_dec_0_up_conv_relu_w,d_dec_0_upsample,d_dec_0_up_conv_relu);
    concatenation<<<dimGrid_dec_0_conc, dimBlock>>>(DEC_0_UP_CONV_RELU_OUTPUT_FEATURES,DEC_0_UP_CONV_RELU_N, d_dec_0_concatenate, d_enc_3_conv_relu_1,d_dec_0_up_conv_relu);
    conv_relu<<<dimGrid_dec_00, dimBlock>>>(DEC_0_CONV_RELU_0_OUTPUT_FEATURES,DEC_0_CONV_RELU_0_N,DEC_0_CONV_RELU_0_K,DEC_0_CONV_RELU_0_INPUT_FEATURES,d_dec_0_conv_relu_0_w,d_dec_0_concatenate,d_dec_0_conv_relu_0);
    conv_relu<<<dimGrid_dec_01, dimBlock>>>(DEC_0_CONV_RELU_1_OUTPUT_FEATURES,DEC_0_CONV_RELU_1_N,DEC_0_CONV_RELU_1_K,DEC_0_CONV_RELU_1_INPUT_FEATURES,d_dec_0_conv_relu_1_w,d_dec_0_conv_relu_0,d_dec_0_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 1--------------------------------------
    upsampling<<<dimGrid_dec_1_up, dimBlock>>>(DEC_1_UP_CONV_RELU_INPUT_FEATURES, DEC_1_UP_CONV_RELU_N, DEC_0_CONV_RELU_1_OUTPUT_FEATURES, d_dec_1_upsample, d_dec_0_conv_relu_1);
    conv_relu<<<dimGrid_dec_1_up_conv, dimBlock>>>(DEC_1_UP_CONV_RELU_OUTPUT_FEATURES,DEC_1_UP_CONV_RELU_N,DEC_1_UP_CONV_RELU_K,DEC_1_UP_CONV_RELU_INPUT_FEATURES,d_dec_1_up_conv_relu_w,d_dec_1_upsample,d_dec_1_up_conv_relu);
    concatenation<<<dimGrid_dec_1_conc, dimBlock>>>(DEC_1_UP_CONV_RELU_OUTPUT_FEATURES,DEC_1_UP_CONV_RELU_N, d_dec_1_concatenate, d_enc_2_conv_relu_1,d_dec_1_up_conv_relu);
    conv_relu<<<dimGrid_dec_10, dimBlock>>>(DEC_1_CONV_RELU_0_OUTPUT_FEATURES,DEC_1_CONV_RELU_0_N,DEC_1_CONV_RELU_0_K,DEC_1_CONV_RELU_0_INPUT_FEATURES,d_dec_1_conv_relu_0_w,d_dec_1_concatenate,d_dec_1_conv_relu_0);
    conv_relu<<<dimGrid_dec_11, dimBlock>>>(DEC_1_CONV_RELU_1_OUTPUT_FEATURES,DEC_1_CONV_RELU_1_N,DEC_1_CONV_RELU_1_K,DEC_1_CONV_RELU_1_INPUT_FEATURES,d_dec_1_conv_relu_1_w,d_dec_1_conv_relu_0,d_dec_1_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 2--------------------------------------
    upsampling<<<dimGrid_dec_2_up, dimBlock>>>(DEC_2_UP_CONV_RELU_INPUT_FEATURES, DEC_2_UP_CONV_RELU_N, DEC_1_CONV_RELU_1_OUTPUT_FEATURES, d_dec_2_upsample, d_dec_1_conv_relu_1);
    conv_relu<<<dimGrid_dec_2_up_conv, dimBlock>>>(DEC_2_UP_CONV_RELU_OUTPUT_FEATURES,DEC_2_UP_CONV_RELU_N,DEC_2_UP_CONV_RELU_K,DEC_2_UP_CONV_RELU_INPUT_FEATURES,d_dec_2_up_conv_relu_w,d_dec_2_upsample,d_dec_2_up_conv_relu);
    concatenation<<<dimGrid_dec_2_conc, dimBlock>>>(DEC_2_UP_CONV_RELU_OUTPUT_FEATURES,DEC_2_UP_CONV_RELU_N, d_dec_2_concatenate, d_enc_1_conv_relu_1,d_dec_2_up_conv_relu);
    conv_relu<<<dimGrid_dec_20, dimBlock>>>(DEC_2_CONV_RELU_0_OUTPUT_FEATURES,DEC_2_CONV_RELU_0_N,DEC_2_CONV_RELU_0_K,DEC_2_CONV_RELU_0_INPUT_FEATURES,d_dec_2_conv_relu_0_w,d_dec_2_concatenate,d_dec_2_conv_relu_0);
    conv_relu<<<dimGrid_dec_21, dimBlock>>>(DEC_2_CONV_RELU_1_OUTPUT_FEATURES,DEC_2_CONV_RELU_1_N,DEC_2_CONV_RELU_1_K,DEC_2_CONV_RELU_1_INPUT_FEATURES,d_dec_2_conv_relu_1_w,d_dec_2_conv_relu_0,d_dec_2_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //-----------------------------DECODER 3--------------------------------------
    upsampling<<<dimGrid_dec_3_up, dimBlock>>>(DEC_3_UP_CONV_RELU_INPUT_FEATURES, DEC_3_UP_CONV_RELU_N, DEC_2_CONV_RELU_1_OUTPUT_FEATURES, d_dec_3_upsample, d_dec_2_conv_relu_1);
    conv_relu<<<dimGrid_dec_3_up_conv, dimBlock>>>(DEC_3_UP_CONV_RELU_OUTPUT_FEATURES,DEC_3_UP_CONV_RELU_N,DEC_3_UP_CONV_RELU_K,DEC_3_UP_CONV_RELU_INPUT_FEATURES,d_dec_3_up_conv_relu_w,d_dec_3_upsample,d_dec_3_up_conv_relu);
    concatenation<<<dimGrid_dec_3_conc, dimBlock>>>(DEC_3_UP_CONV_RELU_OUTPUT_FEATURES,DEC_3_UP_CONV_RELU_N, d_dec_3_concatenate, d_enc_0_conv_relu_1,d_dec_3_up_conv_relu);
    conv_relu<<<dimGrid_dec_30, dimBlock>>>(DEC_3_CONV_RELU_0_OUTPUT_FEATURES,DEC_3_CONV_RELU_0_N,DEC_3_CONV_RELU_0_K,DEC_3_CONV_RELU_0_INPUT_FEATURES,d_dec_3_conv_relu_0_w,d_dec_3_concatenate,d_dec_3_conv_relu_0);
    conv_relu<<<dimGrid_dec_31, dimBlock>>>(DEC_3_CONV_RELU_1_OUTPUT_FEATURES,DEC_3_CONV_RELU_1_N,DEC_3_CONV_RELU_1_K,DEC_3_CONV_RELU_1_INPUT_FEATURES,d_dec_3_conv_relu_1_w,d_dec_3_conv_relu_0,d_dec_3_conv_relu_1);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------

    //----------------------------FINAL LAYER--------------------------------------
    conv_relu_last_layer<<<dimGrid_fin, dimBlock>>>(FINAL_CONV_OUTPUT_FEATURES,FINAL_CONV_N,FINAL_CONV_K,FINAL_CONV_INPUT_FEATURES,d_final_conv_w,d_dec_3_conv_relu_1,d_final_conv);
    Softmax<<<1,FINAL_CONV_N>>>(d_final_conv, d_y);
    //checkCudaError(__LINE__);
    //----------------------------------------------------------------------------
  }
  hipDeviceSynchronize();
  
  time2 = clock()-time2;

  hipMemcpy(y, d_y, N*N_STATES * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  //checkCudaError(__LINE__);

  time1 = clock()-time1;
	printf("Time required for memory and parameters initialization: %.5f seconds\n", ((double)time1-time2) / CLOCKS_PER_SEC);
  printf("Elapsed computation time: %.5f seconds\n", ((double)time2) / CLOCKS_PER_SEC);
  printf("\nTotale elapsed time: %.5f seconds\n\n", ((double)(time1)) / CLOCKS_PER_SEC);

  //hipProfilerStop();

  //VALIDATION
  /*for(int j=0; j<N; j++){
      for(int k=0; k<N_STATES; k++){
          fscanf(pythonOutput,"%f",&value); 
          
          if(y[j*N_STATES+k]!=value){
              printf("ROW: %d COL: %d - %.10f %.10f - abs_err:  %.10f \n",j,k,y[j*N_STATES+k],value, abs(y[j*N_STATES+k]-value));
          }
      }
  }*/

  hipFree(d_enc_0_conv_relu_0);
  hipFree(d_enc_0_conv_relu_1);
  hipFree(d_enc_0_maxpool);

  hipFree(d_enc_1_conv_relu_0);
  hipFree(d_enc_1_conv_relu_1);
  hipFree(d_enc_1_maxpool);

  hipFree(d_enc_2_conv_relu_0);
  hipFree(d_enc_2_conv_relu_1);
  hipFree(d_enc_2_maxpool);

  hipFree(d_enc_3_conv_relu_0);
  hipFree(d_enc_3_conv_relu_1);
  hipFree(d_enc_3_maxpool);

  hipFree(d_central_conv_relu_0);
  hipFree(d_central_conv_relu_1);

  hipFree(d_dec_0_upsample);
  hipFree(d_dec_0_up_conv_relu);
  hipFree(d_dec_0_concatenate);
  hipFree(d_dec_0_conv_relu_0);
  hipFree(d_dec_0_conv_relu_1);

  hipFree(d_dec_1_upsample);
  hipFree(d_dec_1_up_conv_relu);
  hipFree(d_dec_1_concatenate);
  hipFree(d_dec_1_conv_relu_0);
  hipFree(d_dec_1_conv_relu_1);

  hipFree(d_dec_2_upsample);
  hipFree(d_dec_2_up_conv_relu);
  hipFree(d_dec_2_concatenate);
  hipFree(d_dec_2_conv_relu_0);
  hipFree(d_dec_2_conv_relu_1);

  hipFree(d_dec_3_upsample);
  hipFree(d_dec_3_up_conv_relu);
  hipFree(d_dec_3_concatenate);
  hipFree(d_dec_3_conv_relu_0);
  hipFree(d_dec_3_conv_relu_1);

  hipFree(d_final_conv);

  hipFree(d_x);
  hipFree(d_enc_0_conv_relu_0_w);
  hipFree(d_enc_0_conv_relu_1_w);

  hipFree(d_enc_1_conv_relu_0_w);
  hipFree(d_enc_1_conv_relu_1_w);

  hipFree(d_enc_2_conv_relu_0_w);
  hipFree(d_enc_2_conv_relu_1_w);

  hipFree(d_enc_3_conv_relu_0_w);
  hipFree(d_enc_3_conv_relu_1_w);

  hipFree(d_central_conv_relu_0_w);
  hipFree(d_central_conv_relu_1_w);

  hipFree(d_dec_0_up_conv_relu_w);
  hipFree(d_dec_0_conv_relu_0_w);
  hipFree(d_dec_0_conv_relu_1_w);

  hipFree(d_dec_1_up_conv_relu_w);
  hipFree(d_dec_1_conv_relu_0_w);
  hipFree(d_dec_1_conv_relu_1_w);

  hipFree(d_dec_2_up_conv_relu_w);
  hipFree(d_dec_2_conv_relu_0_w);
  hipFree(d_dec_2_conv_relu_1_w);

  hipFree(d_dec_3_up_conv_relu_w);
  hipFree(d_dec_3_conv_relu_0_w);
  hipFree(d_dec_3_conv_relu_1_w);

  hipFree(d_final_conv_w);
  hipFree(d_y);
  //checkCudaError(__LINE__);

  free(y);
  free(final_conv);

  //fclose(pythonOutput);
  return 0;
}