#include "hip/hip_runtime.h"
#include "functions.cuh"

/*
Softmax implementation
  Args:
    x - Input array to perform softmax
    y - Array to save the softmax resultant values
*/
__global__ void Softmax(datatype *x, datatype *y) {
	int index = blockIdx.x*blockDim.x+threadIdx.x; //0*64+0...63

  datatype expx[N_STATES];
  datatype expsum = 0;

  for (int i = 0; i < N_STATES; i++) {
    expx[i] = exp((x+index*N_STATES)[i]);
    expsum += expx[i];
  }

  // To prevent division by zero errors, add EPSILON if expsum is zero
  if (expsum == 0) {
    expsum = EPSILON;
  }

  for (int i = 0; i < N_STATES; i++) {
    (y+index*N_STATES)[i] = expx[i] / expsum;
  }
}

/*
Argmax implementation
  Args:
    x - Input array to perform argmax
    y - Array to save the argmax resultant values
*/
__global__ void Argmax(datatype *x, datatype *y) {
	int index = blockIdx.x*blockDim.x+threadIdx.x;
  #ifdef FLOAT
  datatype maxvalue = FLT_MIN;
  #endif
  #ifdef DOUBLE
  datatype maxvalue = DBL_MIN;
  #endif 
  int maxindex = 0;

  for (int i = 0; i < N_STATES; i++) {
    if ((x+index*N_STATES)[i] > maxvalue) {
      maxvalue = (x+index*N_STATES)[i];
      maxindex = i;
    }
  }

  for (int i = 0; i < N_STATES; i++) {
    if (i == maxindex) {
      (y+index*N_STATES)[i] = 1;
    } else {
      (y+index*N_STATES)[i] = 0;
    }
  }
}

__host__ void checkCudaError(int line) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cuda error: %s. Line: %d\n", hipGetErrorString(err), line);
		exit(-1);
	}
}

__global__ void conv_relu(int conv_relu_output_features, int conv_relu_n, int conv_relu_k, int conv_relu_input_features, datatype *d_weights, datatype *d_input, datatype *d_output){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
  int k = blockIdx.y*blockDim.y+threadIdx.y;
  
  if((k<conv_relu_output_features)&&(i<conv_relu_n)){
    datatype acc = 0;
    int l_min, l_max;

    // Calculate the auxiliary positions respect to the input
    l_min = 0 > (i - conv_relu_k / 2) ? 0 : (i - conv_relu_k / 2); //max
    l_max = (conv_relu_n) < (i + conv_relu_k / 2 + 1) ? (conv_relu_n) : (i + conv_relu_k / 2 + 1); //min

    for (int l = l_min; l < l_max; l++) {
      for (int j = 0; j < conv_relu_input_features; j++) {
        acc += d_input[l*conv_relu_input_features+j] * d_weights[k*conv_relu_k*conv_relu_input_features+(l-i+conv_relu_k/2)*conv_relu_input_features+j]; // Multiply the input and the weight

      }
    }
    d_output[i*conv_relu_output_features+k] = acc < 0 ? 0 : acc; // Relu
  }
}

__global__ void conv_relu_last_layer(int conv_relu_output_features, int conv_relu_n, int conv_relu_k, int conv_relu_input_features, datatype *d_weights, datatype *d_input, datatype *d_output){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
  int k = blockIdx.y*blockDim.y+threadIdx.y;

  if((k<conv_relu_output_features)&&(i<conv_relu_n)){
    datatype acc = 0;
    int l_min, l_max;

    // Calculate the auxiliary positions respect to the input
    l_min = 0 > (i - conv_relu_k / 2) ? 0 : (i - conv_relu_k / 2); //max
    l_max = (conv_relu_n) < (i + conv_relu_k / 2 + 1) ? (conv_relu_n) : (i + conv_relu_k / 2 + 1); //min

    for (int l = l_min; l < l_max; l++) {
      for (int j = 0; j < conv_relu_input_features; j++) {
        acc += d_input[l*conv_relu_input_features+j] * d_weights[k*conv_relu_k*conv_relu_input_features+(l-i+conv_relu_k/2)*conv_relu_input_features+j]; // Multiply the input and the weight
      }
    }
    d_output[i*conv_relu_output_features+k] = acc;
  }
}

__global__ void maxpooling(int enc_conv_relu_output_features, int enc_conv_relu_n, datatype *d_maxpool_output, datatype *d_input_from_conv_rel){
  int k = blockIdx.x*blockDim.x+threadIdx.x;
  int i = blockIdx.y*blockDim.y+threadIdx.y;
  
  if((k<enc_conv_relu_output_features)&&(i<enc_conv_relu_n / 2)){
    d_maxpool_output[i*enc_conv_relu_output_features+k] = d_input_from_conv_rel[(2 * i)*enc_conv_relu_output_features+k] > d_input_from_conv_rel[(2 * i + 1)*enc_conv_relu_output_features+k] ? d_input_from_conv_rel[(2 * i)*enc_conv_relu_output_features+k] : d_input_from_conv_rel[(2 * i + 1)*enc_conv_relu_output_features+k]; //max
    
  }
}

__global__ void upsampling(int dec_up_conv_relu_input_features, int dec_up_conv_relu_n, int dim_conv_relu_input, datatype *d_dec_upsample, datatype *d_conv_relu_input){
  int k = blockIdx.x*blockDim.x+threadIdx.x;
  int i = blockIdx.y*blockDim.y+threadIdx.y;

  if((k<dec_up_conv_relu_input_features)&&(i<(dec_up_conv_relu_n / 2))){
    d_dec_upsample[(2 * i)*dec_up_conv_relu_input_features+k] = d_conv_relu_input[i*dim_conv_relu_input+k];
    d_dec_upsample[(2 * i + 1)*dec_up_conv_relu_input_features+k] = d_conv_relu_input[i*dim_conv_relu_input+k];
  }
}

__global__ void concatenation(int dec_up_conv_relu_output_features, int dec_up_conv_relu_n, datatype *d_dec_concatenate, datatype *d_enc_conv_relu, datatype *d_dec_up_conv_relu){
  int k = blockIdx.x*blockDim.x+threadIdx.x;
  int i = blockIdx.y*blockDim.y+threadIdx.y;

  if((k<dec_up_conv_relu_output_features)&&(i<dec_up_conv_relu_n)){
    d_dec_concatenate[i*dec_up_conv_relu_output_features*2+k] = d_enc_conv_relu[i*dec_up_conv_relu_output_features+k];
    d_dec_concatenate[i*dec_up_conv_relu_output_features*2+(k + dec_up_conv_relu_output_features)] = d_dec_up_conv_relu[i*dec_up_conv_relu_output_features+k];
  }
}