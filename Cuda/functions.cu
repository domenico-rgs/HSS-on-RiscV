#include "hip/hip_runtime.h"
#include "functions.cuh"

/*
Softmax implementation
  Args:
    x - Input array to perform softmax
    y - Array to save the softmax resultant values
*/
__global__ void Softmax(float *x, float *y) {
	int index = blockIdx.x*blockDim.x+threadIdx.x;

  float expx[N_STATES];
  float expsum = 0;

  for (int i = 0; i < N_STATES; i++) {
    expx[i] = exp(x[index+i]);
    expsum += expx[i];
  }

  // To prevent division by zero errors, add EPSILON if expsum is zero
  if (expsum == 0) {
    expsum = EPSILON;
  }

  for (int i = 0; i < N_STATES; i++) {
    y[index+i] = expx[i] / expsum;
  }
}

/*
Argmax implementation
  Args:
    x - Input array to perform argmax
    y - Array to save the argmax resultant values
*/
__global__ void Argmax(float *x, float *y) {
	int index = blockIdx.x*blockDim.x+threadIdx.x;

  float maxvalue = __FLT_MIN__;
  int maxindex = 0;

  for (int i = 0; i < N_STATES; i++) {
    if (x[index+i] > maxvalue) {
      maxvalue = x[index+i];
      maxindex = i;
    }
  }

  for (int i = 0; i < N_STATES; i++) {
    if (i == maxindex) {
      y[index+i] = 1;
    } else {
      y[index+i] = 0;
    }
  }
}

__host__ void checkCudaError(int line) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cuda error: %s. Line: %d\n", hipGetErrorString(err), line);
		exit(-1);
	}
}

__global__ void conv_relu(int conv_relu_output_features, int conv_relu_n, int conv_relu_k, int conv_relu_input_features, float *d_weights, float *d_input, float *d_output){
  int k = blockIdx.y*blockDim.y+threadIdx.y;
	int i = blockIdx.x*blockDim.x+threadIdx.x;

  float acc = 0;
  int l_min, l_max;

  if((k<conv_relu_output_features)&&(i<conv_relu_n)){
    // Calculate the auxiliary positions respect to the input
    l_min = max(0, i - conv_relu_k / 2);
    l_max = min(conv_relu_n, i + conv_relu_k / 2 + 1);

    for (int l = l_min; l < l_max; l++) {
      for (int j = 0; j < conv_relu_input_features; j++) {
        acc += d_input[l*conv_relu_input_features+j] * d_weights[k*conv_relu_k*conv_relu_input_features+(l-i+conv_relu_k/2)*conv_relu_input_features+j]; // Multiply the input and the weight
      }
    }
    d_output[i*conv_relu_output_features+k] = ReLU(acc); // Save the accumulator value
  }
}

__global__ void maxpooling(int enc_conv_relu_output_features, int enc_conv_relu_n, float *d_maxpool_output, float *d_input_from_conv_rel){
  int k = blockIdx.y*blockDim.y+threadIdx.y;
	int i = blockIdx.x*blockDim.x+threadIdx.x;

  if((k<enc_conv_relu_output_features)&&(i<enc_conv_relu_n / 2)){
    d_maxpool_output[i*enc_conv_relu_output_features+k] = max(d_input_from_conv_rel[(2 * i)*enc_conv_relu_output_features+k], d_input_from_conv_rel[(2 * i + 1)*enc_conv_relu_output_features+k]);
  }
}

__global__ void upsampling(int dec_up_conv_relu_input_features, int dec_up_conv_relu_n, float *d_dec_upsample, float *d_conv_relu_input){
  int k = blockIdx.y*blockDim.y+threadIdx.y;
	int i = blockIdx.x*blockDim.x+threadIdx.x;

  if((k<dec_up_conv_relu_input_features)&&(i<dec_up_conv_relu_n / 2)){
    d_dec_upsample[(2 * i)*dec_up_conv_relu_input_features+k] = d_conv_relu_input[i*dec_up_conv_relu_input_features+k];
    d_dec_upsample[(2 * i + 1)*dec_up_conv_relu_input_features+k] = d_conv_relu_input[i*dec_up_conv_relu_input_features+k];
  }
}

__global__ void concatenation(int dec_up_conv_relu_output_features, int dec_up_conv_relu_n, float *d_dec_concatenate, float *d_enc_conv_relu, float *d_dec_up_conv_relu){
  int k = blockIdx.y*blockDim.y+threadIdx.y;
	int i = blockIdx.x*blockDim.x+threadIdx.x;

  if((k<dec_up_conv_relu_output_features)&&(i<dec_up_conv_relu_n)){
    d_dec_concatenate[i*dec_up_conv_relu_output_features+k] = d_enc_conv_relu[i*dec_up_conv_relu_output_features+k];
    d_dec_concatenate[i*dec_up_conv_relu_output_features+(k + dec_up_conv_relu_output_features)] = d_dec_up_conv_relu[i*dec_up_conv_relu_output_features+k];
  }
}